#include "hip/hip_runtime.h"


#include <iostream>
#include <time.h>
#include <omp.h>
#include <fstream>
#include <sstream>
#include <vector>
extern "C" {
float* gpgpu_correlation_mat(float** signals, int n, int signal_count);
void SplitByBatches(float** currentShiftSignals, int n, int signalCount, int shiftWidth, int batchSize);
void shift_compute(float** fullSignals, int n, int signalCount, int shiftWidth, int batchSize);



    __global__ void correlation(float *x, float *y, float *num, float *denom, unsigned int n, float avg_x, float avg_y)
    {
        unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;
        unsigned int stride = blockDim.x*gridDim.x;

        __shared__ float sum_x[256];
        __shared__ float sum_y[256];
        __shared__ float sum_pairwise[256];


        double temp_x = 0.0;
        double temp_y = 0.0;
        double temp_pairwise = 0.0;

        while(index < n)
        {
            temp_x += (x[index] - avg_x) * (x[index] - avg_x);
            temp_y += (y[index] - avg_y) * (y[index] - avg_y);
            temp_pairwise += (x[index] - avg_x) * (y[index] - avg_y);

            index += stride;
        }

        sum_x[threadIdx.x] = temp_x;
        sum_y[threadIdx.x] = temp_y;
        sum_pairwise[threadIdx.x] = temp_pairwise;

        __syncthreads();

        // reduction
        unsigned int i = blockDim.x/2;
        while(i != 0)
        {
            if(threadIdx.x < i)
            {
                sum_x[threadIdx.x] += sum_x[threadIdx.x + i];
                sum_y[threadIdx.x] += sum_y[threadIdx.x + i];
                sum_pairwise[threadIdx.x] += sum_pairwise[threadIdx.x + i];
            }
            __syncthreads();
            i /= 2;
        }


        if(threadIdx.x == 0)
        {
            atomicAdd(denom, sqrtf(sum_x[0] * sum_y[0]));
            atomicAdd(num, sum_pairwise[0]);
        }
    }

    double mean(float* data, int n)
    {
        double sum = 0;
        #pragma omp parallel for schedule(static) reduction (+:sum)
        for(int i = 0; i < n; i++)
        {
            sum += data[i];
        }
        return sum / n;
    }



    void SplitByBatches(float** currentShiftSignals, int n, int signalCount, int shiftWidth, int batchSize)
    {
        float** batch = (float**)malloc(signalCount*sizeof(float*));
        for(int k = 0; k < signalCount; k++)
        {
            batch[k] = (float*)malloc(batchSize * sizeof(float));
        }
        

        for (int batchIndex = 0; batchIndex < n - batchSize; batchIndex += batchSize)
        {
            printf("Batch %i\n", batchIndex);
            for(int k = 0; k < signalCount; k++)
            {
                for(int j = 0; j < batchSize; j++)
                {
                    batch[k][j] = currentShiftSignals[k][j + batchIndex];
                }
            }

            float * result = gpgpu_correlation_mat(batch, batchSize, signalCount);

            for(int i = 0; i < signalCount; i++)
            {
                for(int j = 0; j < signalCount; j++)
                {   
                    printf("%.2f\t|\t", result[i * signalCount + j]);
                }
                std::cout << "\n";
            }
           
        }
        //freeing
        for(int k = 0; k < signalCount; k++)
        {
            free(batch[k]);
        }
        free(batch);
    }


    float* gpgpu_correlation_mat(float** signals, int n, int signal_count)
    {
        dim3 gridSize = 256;
        dim3 blockSize = 256;

        float* result = (float*)malloc(signal_count * signal_count * sizeof(float));

        float *denom_res = (float*)malloc(sizeof(float));   
        float *num_res = (float*)malloc(sizeof(float));

        float *d_prod_num, *d_prod_denom;
        float *d_x, *d_y;


        hipMalloc((void**)&d_prod_num, sizeof(float));
        hipMalloc((void**)&d_prod_denom, sizeof(float));
        hipMalloc((void**)&d_x, n*sizeof(float));
        hipMalloc((void**)&d_y, n*sizeof(float));

        float x_mean, y_mean;

        for(int i = 0; i < signal_count; i++)
        {
            for(int j = 0; j < signal_count; j++)
            {   
                hipMemset(d_prod_num, 0.0f, sizeof(float));
                hipMemset(d_prod_denom, 0.0f, sizeof(float));
                hipMemcpyAsync(d_x, (void*)signals[i], n*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpyAsync(d_y, (void*)signals[j], n*sizeof(float), hipMemcpyHostToDevice);

                x_mean = mean(signals[i], n);
                y_mean = mean(signals[j], n);

                correlation<<<gridSize, blockSize>>>(d_x, d_y, d_prod_num, d_prod_denom, n, x_mean, y_mean);
                hipMemcpyAsync(num_res, d_prod_num, sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpyAsync(denom_res, d_prod_denom, sizeof(float), hipMemcpyDeviceToHost);

                result[i * signal_count + j] = (*num_res) / (*denom_res);
            }
        }

        hipFree(d_prod_num);
        hipFree(d_prod_denom);
        hipFree(d_x);
        hipFree(d_y);

        free(denom_res);
        free(num_res);

        return result;
    }
}




int main(int argc, char** argv)
{
    std::ifstream f;
       
    f.open ("ClosedEyes.asc");

    std::string line, val;                  
    std::vector<std::vector<float>> array;    

    while (std::getline (f, line)) {      
    std::vector<float> v;                 
    std::stringstream s (line);         
    while (getline (s, val, ' '))       
        v.push_back (std::stof (val));  
    array.push_back (v);                
    }

    unsigned int n = array.size();
    int signal_count = array[0].size();
    

    float** h_x = (float**)malloc(signal_count*sizeof(float*));

    for(int i = 0; i < signal_count; i++)
    {
        h_x[i] = (float*)malloc(n * sizeof(float));
    }

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < signal_count; j++)
        {
            h_x[j][i] = array[i][j];
        }
    }


    SplitByBatches(h_x, n, signal_count, 100, 100);


    system("pause");

}